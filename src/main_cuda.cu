#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <algorithm>
#include <rays.h>

#include <constants.h>

#define SDL_MAIN_HANDLED
#include <SDL.h>

uint8_t init();

SDL_Window *g_window = NULL;
SDL_Renderer *renderer = NULL;

uint8_t init() {
    uint8_t success = 1;

    if (SDL_Init(SDL_INIT_VIDEO) < 0) {
        printf("No se pudo inicializar SDL: %s\n", SDL_GetError());
        return 0;
    }

    g_window = SDL_CreateWindow("Ray casting", SDL_WINDOWPOS_UNDEFINED,
        SDL_WINDOWPOS_UNDEFINED, SCREEN_WIDTH, SCREEN_HEIGHT, SDL_WINDOW_SHOWN);
    if (!g_window) {
        printf("No se pudo crear la ventana: %s", SDL_GetError());
        return 0;
    }

    return 1;
}
__device__ inline void check_impact_c(sphere *s, float x, float y, float z, int* F) {
    float xd = x - (s->x);
    float yd = y - (s->y);
    float zd = z - (s->z);

    float d = (xd*xd + yd*yd + zd*zd);
    float r = (s->radius) * (s->radius);

    *F = (int)(d <= r);

    return;
}


__device__ void ray_marching_c(ray *r, sphere *spheres, int n_spheres, color *c) {
    do {
        for (int i = 0; i < n_spheres; i++) {
            int F;
            check_impact_c(&spheres[i], r->x, r->y, r->z, &F);

            if (F) {
                float dz = (r->z);
                r->intensity = (r->k)/(dz*dz);
                c->R = min(spheres[i].R * r->intensity, 255.);
                c->G = min(spheres[i].G * r->intensity, 255.);
                c->B = min(spheres[i].B * r->intensity, 255.);
                return;
            }
        }

        r->z += 1;
        r->alive = r->z < 100;
    } while(r->alive);
}

sphere* initialize_spheres(int n_spheres) {
    sphere *s;
    s = (sphere *)malloc(sizeof(sphere)*n_spheres);

    // initialize random seed
    time_t t;
    srand((unsigned) time(&t));

    for (int i = 0; i < n_spheres; i++) {
        s[i].x = rand() % SCREEN_WIDTH;
        s[i].y = rand() % SCREEN_HEIGHT;
        s[i].z = rand() % SCREEN_WIDTH + 50;
        // s[0]->z = 10.;

        s[i].R = rand() % 255;
        s[i].G = rand() % 255;
        s[i].B = rand() % 255;

        s[i].radius = s[i].z - s[i].z*0.1;
    }

    return s;
}

__global__ void sequential_render(int8_t *pixels, sphere *spheres, int n_spheres, ray *rays, color *colors) {
    int tid= threadIdx.x + blockDim.x * blockIdx.x;
    const int x = tid % (SCREEN_WIDTH);
    const int y = tid / (SCREEN_WIDTH);

    const int c_i = tid*4;

    if (tid < SCREEN_WIDTH * SCREEN_HEIGHT) {
      ray *r = &rays[tid];
      r->x = x;
      r->y = y;
      r->z = 0.;
      r->k = 5001.;
      r->alive = 1;
      color *c = &colors[tid];
      c->R = 0; c->B = 0; c->G=0;
      ray_marching_c(r, spheres, n_spheres, c);
      pixels[c_i + 0] = c->R;
      pixels[c_i + 1] = c->G;
      pixels[c_i + 2] = c->B;
      pixels[c_i + 3] = SDL_ALPHA_OPAQUE;

    }
}

struct arr{
    SDL_Renderer *renderer;
    SDL_Texture *texture;
    int8_t *pixels;
};

void foo(void* userData){
    struct arr *b = (struct arr*)userData;
    SDL_UpdateTexture(b->texture, NULL, b->pixels, SCREEN_WIDTH * 4);
    SDL_RenderCopy(b->renderer, b->texture, NULL, NULL);
    SDL_RenderPresent(b->renderer);
    return;
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Debe ingresar la cantidad de esferas\n");
        return EXIT_FAILURE;
    }

    SDL_SetMainReady();

    int n_spheres = atoi(argv[1]);

    if (!init()) {
        printf("No se pudo inicializar\n");
        return EXIT_FAILURE;
    }

    SDL_Renderer *renderer = SDL_CreateRenderer(g_window, -1,
        SDL_RENDERER_SOFTWARE);

    // Texture used for the 3d world simulation
    SDL_Texture *texture = SDL_CreateTexture(renderer,
            SDL_PIXELFORMAT_ABGR8888, SDL_TEXTUREACCESS_STREAMING,
            SCREEN_WIDTH, SCREEN_HEIGHT);

    int8_t *pixels = (int8_t *)malloc(
        SCREEN_WIDTH * SCREEN_HEIGHT * 4 * sizeof(int8_t));
    
    int8_t *pixels_dev;
    
    hipMalloc((void **)&pixels_dev, SCREEN_WIDTH * SCREEN_HEIGHT * 4 * sizeof(int8_t));

    uint8_t running = 1;
    SDL_Event event;

    sphere *spheres = initialize_spheres(n_spheres);

    sphere *spheres_dev;
    hipMalloc((void **)&spheres_dev, n_spheres*sizeof(sphere));
    hipMemcpy(spheres_dev,spheres,n_spheres*sizeof(sphere), hipMemcpyHostToDevice);
    free(spheres);

    // alloc rays & colors
    color *colors;
    ray *rays;
    hipMalloc((void **)&colors, SCREEN_WIDTH * SCREEN_HEIGHT * sizeof(color));
    hipMalloc((void **)&rays, SCREEN_WIDTH * SCREEN_HEIGHT * sizeof(ray));

    uint32_t iters = 0;
    int block_size = 256, grid_size = (int)ceil((float)SCREEN_HEIGHT*SCREEN_WIDTH/256.);
    float secs = 0;
    SDL_SetRenderDrawColor(renderer, 0, 0, 0, SDL_ALPHA_OPAQUE);
    hipStream_t x,y;
    hipStreamCreate(&x);
    hipStreamCreate(&y);
    int off = 0;
    while(running) {
        // hipStream_t us;
        // if (off%2) us = x;
        // else us = y;
        uint64_t start = SDL_GetPerformanceCounter();
        //
        SDL_RenderClear(renderer);

        while(SDL_PollEvent(&event)) {
            if (event.type == SDL_QUIT) {
                running = 0;
                break;
            }
        }
        //hipMemcpy(dest, orig,  sizeof(), hipMemcpyDeviceToHost);
        struct arr *arr;
        arr = new (struct arr);
        arr->pixels = pixels;
        arr->texture = texture;
        arr->renderer = renderer;
        sequential_render<<<grid_size,block_size,0,off%2?x:y>>>(pixels_dev, spheres_dev, n_spheres, rays, colors);
        hipMemcpyAsync(pixels, pixels_dev, SCREEN_WIDTH * SCREEN_HEIGHT * 4 * sizeof(int8_t), hipMemcpyDeviceToHost,off%2?x:y);
        hipLaunchHostFunc(off%2?x:y, (hipHostFn_t)foo, (void*)arr);

        // calculation of frames per second.
        uint64_t end = SDL_GetPerformanceCounter();
        double freq = (double)SDL_GetPerformanceFrequency();
        secs = (float)(end - start) /(freq);
        printf("%f\n", 1/(secs));
        off++;
    }
    printf("%f\n", 1/(secs));

    SDL_DestroyTexture(texture);
    SDL_DestroyRenderer(renderer);
    SDL_DestroyWindow(g_window);

    hipFree(spheres_dev);
    hipFree(pixels_dev);
    hipFree(rays);
    hipFree(colors);

    SDL_Quit();
    return EXIT_SUCCESS;
}
