#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <rays.h>

#include <constants.h>

#define SDL_MAIN_HANDLED
#include <SDL.h>

uint8_t init();

SDL_Window *g_window = NULL;
SDL_Renderer *renderer = NULL;

uint8_t init() {
    uint8_t success = 1;

    if (SDL_Init(SDL_INIT_VIDEO) < 0) {
        printf("No se pudo inicializar SDL: %s\n", SDL_GetError());
        return 0;
    }

    g_window = SDL_CreateWindow("Ray casting", SDL_WINDOWPOS_UNDEFINED,
        SDL_WINDOWPOS_UNDEFINED, SCREEN_WIDTH, SCREEN_HEIGHT, SDL_WINDOW_SHOWN);
    if (!g_window) {
        printf("No se pudo crear la ventana: %s", SDL_GetError());
        return 0;
    }

    return 1;
}
__device__ inline void check_impact_c(sphere *s, float x, float y, float z, int* F) {
    float xd = x - (s->x);
    float yd = y - (s->y);
    float zd = z - (s->z);

    float d = (xd*xd + yd*yd + zd*zd);
    float r = (s->radius) * (s->radius);

    *F = (int)(d <= r);

    return;
}


__device__ void ray_marching_c(ray *r, sphere *spheres, int n_spheres, color *c) {
    do {
        for (int i = 0; i < n_spheres; i++) {
            int F;
            check_impact_c(&spheres[i], r->x, r->y, r->z, &F);

            if (F) {
                float dz = (r->z);
                r->intensity = (r->k)/(dz*dz);
                c->R = min(spheres[i].R * r->intensity, 255.);
                c->G = min(spheres[i].G * r->intensity, 255.);
                c->B = min(spheres[i].B * r->intensity, 255.);
                return;
            }
        }

        r->z += 1;
        r->alive = r->z < 100;
    } while(r->alive);
}

__global__ void sequential_render(int8_t *pixels, sphere *spheres, int n_spheres, ray *rays, color *colors) {
    int tid= threadIdx.x + blockDim.x * blockIdx.x;
    const int x = tid % (SCREEN_WIDTH);
    const int y = tid / (SCREEN_WIDTH);

    const int c_i = tid*4;

    if (tid < SCREEN_WIDTH * SCREEN_HEIGHT) {
      ray *r = &rays[tid];
      r->x = x;
      r->y = y;
      r->z = 0.;
      r->k = 5001.;
      r->alive = 1;
      color *c = &colors[tid];
      c->R = 0; c->B = 0; c->G=0;
      ray_marching_c(r, spheres, n_spheres, c);
      pixels[c_i + 0] = c->R;
      pixels[c_i + 1] = c->G;
      pixels[c_i + 2] = c->B;
      pixels[c_i + 3] = SDL_ALPHA_OPAQUE;

    }
}

struct arr{
    SDL_Renderer *renderer;
    SDL_Texture *texture;
    sphere *spheres;
    int n_spheres;
    int8_t *pixels;
};

void foo(void* userData){
    struct arr *b = (struct arr*)userData;
    move_spheres(b->spheres, b->n_spheres);
    SDL_RenderClear(b->renderer);
    SDL_UpdateTexture(b->texture, NULL, b->pixels, SCREEN_WIDTH * 4);
    SDL_RenderCopy(b->renderer, b->texture, NULL, NULL);
    SDL_RenderPresent(b->renderer);
    return;
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Debe ingresar la cantidad de esferas\n");
        return EXIT_FAILURE;
    }

    SDL_SetMainReady();

    int n_spheres = atoi(argv[1]);

    if (!init()) {
        printf("No se pudo inicializar\n");
        return EXIT_FAILURE;
    }

    SDL_Renderer *renderer = SDL_CreateRenderer(g_window, -1,
        SDL_RENDERER_SOFTWARE);

    // Texture used for the 3d world simulation
    SDL_Texture *texture = SDL_CreateTexture(renderer,
            SDL_PIXELFORMAT_ABGR8888, SDL_TEXTUREACCESS_STREAMING,
            SCREEN_WIDTH, SCREEN_HEIGHT);

    int8_t *pixels;
    hipHostMalloc(&pixels,
        SCREEN_WIDTH * SCREEN_HEIGHT * 4 * sizeof(int8_t));
    
    int8_t *pixels_dev;
    
    hipMalloc((void **)&pixels_dev, SCREEN_WIDTH * SCREEN_HEIGHT * 4 * sizeof(int8_t));

    uint8_t running = 1;
    SDL_Event event;

    sphere *spheres = initialize_spheres(n_spheres);
    sphere *spheres_async;
    hipHostMalloc((void**)&spheres_async,n_spheres * sizeof(spheres));

    memcpy(spheres_async, spheres, n_spheres * sizeof(spheres));

    sphere *spheres_dev;
    hipMalloc((void **)&spheres_dev, n_spheres*sizeof(sphere));
    hipMemcpy(spheres_dev,spheres,n_spheres*sizeof(sphere), hipMemcpyHostToDevice);

    // alloc rays & colors
    color *colors;
    ray *rays;
    hipMalloc((void **)&colors, SCREEN_WIDTH * SCREEN_HEIGHT * sizeof(color));
    hipMalloc((void **)&rays, SCREEN_WIDTH * SCREEN_HEIGHT * sizeof(ray));

    uint32_t iters = 0;
    int block_size = 256, grid_size = (int)ceil((float)(SCREEN_HEIGHT*SCREEN_WIDTH)/256.);
    float secs = 0;
    SDL_SetRenderDrawColor(renderer, 0, 0, 0, SDL_ALPHA_OPAQUE);
    hipStream_t x,y;
    hipStreamCreate(&x);
    hipStreamCreate(&y);
    int off = 0;
    struct arr *arr;
    arr = new (struct arr);
    arr->pixels = pixels;
    arr->texture = texture;
    arr->renderer = renderer;
    arr->spheres = spheres_async;
    arr->n_spheres = n_spheres;
    // int size = SCREEN_HEIGHT*SCREEN_WIDTH;
    while(running) {
        // hipStream_t us;
        // if (off%2) us = x;
        // else us = y;
        uint64_t start = SDL_GetPerformanceCounter();
        // //
        // //hipMemcpy(dest, orig,  sizeof(), hipMemcpyDeviceToHost);
        // //SDL_RenderClear(renderer);

        while(SDL_PollEvent(&event)) {
            if (event.type == SDL_QUIT) {
                running = 0;
                break;
            }
        }
        

        /**

        saaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaa
        **/
        sequential_render<<<grid_size,block_size,0,off%2?x:y>>>(pixels_dev, spheres_dev, n_spheres, rays, colors);
        hipMemcpyAsync(spheres_dev,spheres_async,n_spheres*sizeof(sphere), hipMemcpyHostToDevice, off%2?x:y);
        hipMemcpyAsync(pixels, pixels_dev, SCREEN_WIDTH * SCREEN_HEIGHT * 4 * sizeof(int8_t), hipMemcpyDeviceToHost,off%2?x:y);
        hipLaunchHostFunc(off%2?x:y, (hipHostFn_t)foo, (void*)arr);

        

        //hipMemcpy(pixels, pixels_dev, SCREEN_WIDTH * SCREEN_HEIGHT * 4 * sizeof(int8_t), hipMemcpyDeviceToHost);
        //sequential_render<<<grid_size,block_size>>>(pixels_dev, spheres_dev, n_spheres, rays, colors);

        

        // calculation of frames per second.
        uint64_t end = SDL_GetPerformanceCounter();
        double freq = (double)SDL_GetPerformanceFrequency();
        secs = (float)(end - start) /(freq);
        printf("%f\n", 1/(secs));
        off++;
        
    }
    printf("%f\n", 1/(secs));

    SDL_DestroyTexture(texture);
    SDL_DestroyRenderer(renderer);
    SDL_DestroyWindow(g_window);

    hipFree(spheres_dev);
    hipFree(pixels_dev);
    hipFree(rays);
    hipFree(colors);

    SDL_Quit();
    return EXIT_SUCCESS;
}
